#include "hip/hip_runtime.h"
//****************************************************************************************
//
// Author : Aniruddha Shembekar, University of Southern California
//
//****************************************************************************************

#include "string.h"
#include <stdio.h>
#include <cmath>
#include <Eigen/Eigen>
#include "nlopt.hpp"
#include <iostream>
#include <vector>
#include "opt_obj_eigen.hpp"
#include "transformation_utilities.hpp"
#include "utilities.hpp"
#include "file_rw.hpp"
#include <chrono>

int main ()
{
    Eigen::initParallel();
    Eigen::setNbThreads(8);
    std::string file_num = "3";
    std::string file_path = "/home/aniruddha/Desktop/Composite_Layup/CUDA_experiments/data/";
    
    Eigen::MatrixXd No_particles_mat = file_rw::file_read_mat(file_path+"no_particles"+file_num+".csv");
    Eigen::MatrixXd coords = file_rw::file_read_mat(file_path+"coords"+file_num+".csv");
    Eigen::MatrixXd sheet_prop = file_rw::file_read_mat(file_path+"properties"+file_num+".csv");
    Eigen::MatrixXd node_info = file_rw::file_read_mat(file_path+"node_info"+file_num+".csv");
    Eigen::MatrixXd free_indices = file_rw::file_read_mat(file_path+"free_indices"+file_num+".csv");
    Eigen::MatrixXd X_init_mat = file_rw::file_read_mat(file_path+"X_init"+file_num+".csv");

    double OptH = 1e-11;
    double OptXtolRel = 1e-5;
    
    std::vector<double> X_init;

    for (int i=0;i<X_init_mat.cols();++i)
    {
        X_init.push_back(X_init_mat(0,i));
    }

    int No_particles = No_particles_mat(0,0);

    opt_obj::opt_obj OptObj(coords, sheet_prop, node_info, No_particles, free_indices, X_init, OptH, OptXtolRel); 
    
    std::cout << "optm start" << std::endl; 
    auto gpu_start = std::chrono::high_resolution_clock::now();
    
    bool flag = OptObj.solveOPT();
    auto gpu_end = std::chrono::high_resolution_clock::now();
    std::cout << "vector_add_gpu time: " << std::chrono::duration_cast<std::chrono::milliseconds>(gpu_end - gpu_start).count() << " milliseconds.\n";
    
    std::cout << "optm end : " << flag << std::endl; 
    
    std::vector<double> X_f_vec = OptObj.get_solx();
    Eigen::MatrixXd X_f(X_f_vec.size(),1);
    
    for (int i=0;i<X_f_vec.size();++i)
    {
        X_f(i,0) = X_f_vec[0];
    }

    double fmin = OptObj.get_solminf();
    std::cout << fmin << std::endl; 

    std::cout << "everything executed correctly..." << std::endl;

    return 0;
}