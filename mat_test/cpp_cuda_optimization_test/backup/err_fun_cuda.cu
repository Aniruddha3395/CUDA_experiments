#include "hip/hip_runtime.h"
// #include "/usr/local/cuda-9.1/include/hip/hip_runtime.h"
#include "string.h"
#include <stdio.h>
#include <cmath>
#include </usr/local/include/eigen3/Eigen/Eigen>
#include <iostream>
#include <vector>
// #include "transformation_utilities.hpp"
// #include "utilities.hpp"
// #include "file_rw.hpp"
#include <chrono>
#include "err_fun_cuda.h"


__global__ void energy_calc(double *, double *, double *, double *, double *, double, int);

double error_fun_cuda(int no_particles, Eigen::MatrixXd coords, Eigen::MatrixXd properties, Eigen::MatrixXd node_info,
    Eigen::MatrixXd free_indices, std::vector<double> x)
{
    // auto gpu_start = std::chrono::high_resolution_clock::now();
    // std::string file_path = "/home/aniruddha/Downloads/cuda_test/data/";
    // int no_particles = 160;
    // Eigen::MatrixXd coords = file_rw::file_read_mat(file_path+"coords.csv");
    // Eigen::MatrixXd properties = file_rw::file_read_mat(file_path+"properties.csv");
    // Eigen::MatrixXd node_info = file_rw::file_read_mat(file_path+"node_info.csv");
    // Eigen::MatrixXd free_indices = file_rw::file_read_mat(file_path+"free_indices.csv");
    // Eigen::MatrixXd x_mat = file_rw::file_read_mat(file_path+"X_init.csv");
    
    // std::vector<double> x;

    // for (int i=0;i<x_mat.cols();++i)
    // {
    //     x.push_back(x_mat(0,i));
    // }

    
    Eigen::MatrixXd spr_pot = Eigen::MatrixXd::Constant(no_particles,1,0);
    Eigen::MatrixXd shr_pot = Eigen::MatrixXd::Constant(no_particles,1,0);
    Eigen::MatrixXd bndg_pot = Eigen::MatrixXd::Constant(no_particles,1,0);
    int counter = -1;
    
    // for (int i = 0; i < free_indices.rows(); i++)
    // {
    //     coords(free_indices(i,0),0) = x[counter+=1];
    //     coords(free_indices(i,0),1) = x[counter+=1];
    //     coords(free_indices(i,0),2) = x[counter+=1];
    // }

    for ( int i = 0; i < free_indices.rows(); i+=3)
    {
        coords(free_indices(i,0)-1,0) = x[i];
        coords(free_indices(i,0)-1,1) = x[i+1];
        coords(free_indices(i,0)-1,2) = x[i+2];
    }

    double coords_arr[coords.rows()*coords.cols()];
    double *ptr_coords_arr;
    ptr_coords_arr = &coords_arr[0]; 
    Eigen::Map<Eigen::MatrixXd>(ptr_coords_arr,coords.rows(),coords.cols()) = coords;

    double node_info_arr[node_info.rows()*node_info.cols()];
    double *ptr_node_info_arr;
    ptr_node_info_arr = &node_info_arr[0]; 
    Eigen::Map<Eigen::MatrixXd>(ptr_node_info_arr,node_info.rows(),node_info.cols()) = node_info;    

    
    double grav_pot = coords.col(2).array().abs().sum();

    double *ptr_node_info;
    double *ptr_coords;
    double *ptr_spr_pot, *ptr_shr_pot, *ptr_bndg_pot;

    int pot_rows = spr_pot.rows();

    hipMalloc((void**)&ptr_node_info, node_info.rows()*node_info.cols()*sizeof(double));
    hipMalloc((void**)&ptr_coords, coords.rows()*coords.cols()*sizeof(double));
    hipMalloc((void**)&ptr_spr_pot, pot_rows*1*sizeof(double));
    hipMalloc((void**)&ptr_shr_pot, pot_rows*1*sizeof(double));
    hipMalloc((void**)&ptr_bndg_pot, pot_rows*1*sizeof(double));
       
    hipMemcpy(ptr_node_info, node_info_arr, node_info.rows()*node_info.cols()*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(ptr_coords, coords_arr, coords.rows()*coords.cols()*sizeof(double),hipMemcpyHostToDevice);

    dim3 numBlocks(no_particles);
    dim3 threadsPerBlock(1);

    energy_calc <<<numBlocks, threadsPerBlock>>> (ptr_node_info, ptr_coords, ptr_spr_pot, ptr_shr_pot, 
        ptr_bndg_pot, properties(4,0), coords.rows());
 
    hipMemcpy(spr_pot.data(), ptr_spr_pot, pot_rows*1*sizeof(double),hipMemcpyDeviceToHost);
    hipMemcpy(shr_pot.data(), ptr_shr_pot, pot_rows*1*sizeof(double),hipMemcpyDeviceToHost);
    hipMemcpy(bndg_pot.data(), ptr_bndg_pot, pot_rows*1*sizeof(double),hipMemcpyDeviceToHost);
        
    hipFree(ptr_node_info);
    hipFree(ptr_coords);
    hipFree(ptr_spr_pot);
    hipFree(ptr_shr_pot);
    hipFree(ptr_bndg_pot);

    double E = properties(0,0)*grav_pot + properties(1,0)*spr_pot.sum() + properties(2,0)*shr_pot.sum() + properties(3,0)*bndg_pot.sum();
    // std::cout << E << std::endl;
    // auto gpu_end = std::chrono::high_resolution_clock::now();
    // std::cout << "vector_add_gpu time: " << std::chrono::duration_cast<std::chrono::nanoseconds>(gpu_end - gpu_start).count() << " nanoseconds.\n";
    return E;
}

__global__ void energy_calc(double *node_info, double *coords, 
    double *spr_pot, double *shr_pot, double *bndg_pot, double properties_val, int coords_cols)
{
    int i = blockIdx.x;
    if (i<coords_cols)
    {
        double a[3];
        double b[3];
        double c[3];
        double d[3];
        double norm_a;
        double norm_b;
        double norm_c;
        double norm_d;
        
        // Unit vectors and Spring Energy mg(dh)
        if (node_info[i]==1)
        {
            int idx = node_info[i+coords_cols*10];
            a[0] = coords[idx] - coords[i];
            a[1] = coords[idx+coords_cols] - coords[i+coords_cols];
            a[2] = coords[idx+coords_cols*2] - coords[i+coords_cols*2];    

            norm_a = sqrt(a[0]*a[0]+a[1]*a[1]+a[2]*a[2]);
            spr_pot[i] += (norm_a - properties_val)*(norm_a - properties_val);
            a[0] = a[0] / norm_a;
            a[1] = a[1] / norm_a;
            a[2] = a[2] / norm_a;
        }

        if (node_info[i+coords_cols]==1)
        {
            int idx = node_info[i+coords_cols*11];
            b[0] = coords[idx] - coords[i];
            b[1] = coords[idx+coords_cols] - coords[i+coords_cols];
            b[2] = coords[idx+coords_cols*2] - coords[i+coords_cols*2];
            
            norm_b = sqrt(b[0]*b[0]+b[1]*b[1]+b[2]*b[2]);
            spr_pot[i] += (norm_b - properties_val)*(norm_b - properties_val);
            b[0] = b[0] / norm_b;
            b[1] = b[1] / norm_b;
            b[2] = b[2] / norm_b;
        }
            
        if (node_info[i+coords_cols*2]==1)
        {
            int idx = node_info[i+coords_cols*12];
            c[0] = coords[idx] - coords[i];
            c[1] = coords[idx+coords_cols] - coords[i+coords_cols];
            c[2] = coords[idx+coords_cols*2] - coords[i+coords_cols*2];
            
            norm_c = sqrt(c[0]*c[0]+c[1]*c[1]+c[2]*c[2]);
            spr_pot[i] += (norm_c - properties_val)*(norm_c - properties_val);
            c[0] = c[0] / norm_c;
            c[1] = c[1] / norm_c;
            c[2] = c[2] / norm_c;
        }
        
        if (node_info[i+coords_cols*3]==1)
        {
            int idx = node_info[i+coords_cols*13];
            d[0] = coords[idx] - coords[i];
            d[1] = coords[idx+coords_cols] - coords[i+coords_cols];
            d[2] = coords[idx+coords_cols*2] - coords[i+coords_cols*2];

            norm_d = sqrt(d[0]*d[0]+d[1]*d[1]+d[2]*d[2]);
            spr_pot[i] += (norm_d - properties_val)*(norm_d - properties_val);
            d[0] = d[0] / norm_d;
            d[1] = d[1] / norm_d;
            d[2] = d[2] / norm_d;
        }

        // Shear Energy
        if (node_info[i+coords_cols*6]==1)
        {
            double dot_prod = a[0]*c[0] + a[1]*c[1] + a[2]*c[2];
            if (dot_prod < -1.0)
                dot_prod = -1.0;
            if (dot_prod > 1.0)
                dot_prod = 1.0;
            shr_pot[i] += tan(abs(M_PI_2 - acos(dot_prod)));
        }

        if (node_info[i+coords_cols*7]==1)
        {
            double dot_prod = b[0]*c[0] + b[1]*c[1] + b[2]*c[2];
            if (dot_prod < -1.0)
                dot_prod = -1.0;
            if (dot_prod > 1.0)
                dot_prod = 1.0;
            shr_pot[i] += tan(abs(M_PI_2 - acos(dot_prod)));
        }
        
        if (node_info[i+coords_cols*8]==1)
        {
            double dot_prod = b[0]*d[0] + b[1]*d[1] + b[2]*d[2];
            if (dot_prod < -1.0)
                dot_prod = -1.0;
            if (dot_prod > 1.0)
                dot_prod = 1.0;
            shr_pot[i] += tan(abs(M_PI_2 - acos(dot_prod)));
        }
        
        if (node_info[i+coords_cols*9]==1)
        {
            double dot_prod = a[0]*d[0] + a[1]*d[1] + a[2]*d[2];
            if (dot_prod < -1.0)
                dot_prod = -1.0;
            if (dot_prod > 1.0)
                dot_prod = 1.0;
            shr_pot[i] += tan(abs(M_PI_2 - acos(dot_prod)));
        }

        // Bending Energy
        if (node_info[i+coords_cols*4]==1)
        {
            double dot_prod = a[0]*b[0] + a[1]*b[1] + a[2]*b[2];
            if (dot_prod < -1.0)
                dot_prod = -1.0;
            if (dot_prod > 1.0)
                dot_prod = 1.0;
            bndg_pot[i] += tan((M_PI - acos(dot_prod))/2);
        }
        
        if (node_info[i+coords_cols*5]==1)
        {
            double dot_prod = d[0]*c[0] + d[1]*c[1] + d[2]*c[2];
            if (dot_prod < -1.0)
                dot_prod = -1.0;
            if (dot_prod > 1.0)
                dot_prod = 1.0;
            bndg_pot[i] += tan((M_PI - acos(dot_prod))/2);
        }
    }
}
