#include "hip/hip_runtime.h"
// Author   : Aniruddha Shembekar, Research Engineer, University of Southern California

#include <iostream>
#include </usr/local/include/eigen3/Eigen/Eigen>
#include <vector>
#include <string>
#include <cmath>
#include <nlopt.hpp>
#include "stdlib.h"
#include "opt_obj.h"
#include <chrono>

__global__ void energy_calc(double *node_info, double *coords, 
    double *spr_pot, double *shr_pot, double *bndg_pot, double properties_val, int coords_cols)
{
    int i = threadIdx.x + 1000*blockIdx.x;
    
    if (i<coords_cols)
    {
        double a[3];
        double b[3];
        double c[3];
        double d[3];
        double norm_a;
        double norm_b;
        double norm_c;
        double norm_d;
        spr_pot[i] = 0;
        shr_pot[i] = 0;
        bndg_pot[i] = 0;
        double angle;

        // Unit vectors and Spring Energy mg(dh)
        if (node_info[i]==1)
        {
            int idx = node_info[i+coords_cols*10];
            a[0] = coords[idx] - coords[i];
            a[1] = coords[idx+coords_cols] - coords[i+coords_cols];
            a[2] = coords[idx+coords_cols*2] - coords[i+coords_cols*2];    

            norm_a = sqrt(a[0]*a[0]+a[1]*a[1]+a[2]*a[2]);
            spr_pot[i] += (norm_a - properties_val)*(norm_a - properties_val);
            a[0] = a[0] / norm_a;
            a[1] = a[1] / norm_a;
            a[2] = a[2] / norm_a;
        }

        if (node_info[i+coords_cols]==1)
        {
            int idx = node_info[i+coords_cols*11];
            b[0] = coords[idx] - coords[i];
            b[1] = coords[idx+coords_cols] - coords[i+coords_cols];
            b[2] = coords[idx+coords_cols*2] - coords[i+coords_cols*2];
            
            norm_b = sqrt(b[0]*b[0]+b[1]*b[1]+b[2]*b[2]);
            spr_pot[i] += (norm_b - properties_val)*(norm_b - properties_val);
            b[0] = b[0] / norm_b;
            b[1] = b[1] / norm_b;
            b[2] = b[2] / norm_b;
        }
            
        if (node_info[i+coords_cols*2]==1)
        {
            int idx = node_info[i+coords_cols*12];
            c[0] = coords[idx] - coords[i];
            c[1] = coords[idx+coords_cols] - coords[i+coords_cols];
            c[2] = coords[idx+coords_cols*2] - coords[i+coords_cols*2];
            
            norm_c = sqrt(c[0]*c[0]+c[1]*c[1]+c[2]*c[2]);
            spr_pot[i] += (norm_c - properties_val)*(norm_c - properties_val);
            c[0] = c[0] / norm_c;
            c[1] = c[1] / norm_c;
            c[2] = c[2] / norm_c;
        }
        
        if (node_info[i+coords_cols*3]==1)
        {
            int idx = node_info[i+coords_cols*13];
            d[0] = coords[idx] - coords[i];
            d[1] = coords[idx+coords_cols] - coords[i+coords_cols];
            d[2] = coords[idx+coords_cols*2] - coords[i+coords_cols*2];

            norm_d = sqrt(d[0]*d[0]+d[1]*d[1]+d[2]*d[2]);
            spr_pot[i] += (norm_d - properties_val)*(norm_d - properties_val);
            d[0] = d[0] / norm_d;
            d[1] = d[1] / norm_d;
            d[2] = d[2] / norm_d;
        }

        // Shear Energy
        if (node_info[i+coords_cols*6]==1)
        {
            double dot_prod = a[0]*c[0] + a[1]*c[1] + a[2]*c[2];
            if (dot_prod < -1.0)
                dot_prod = -1.0;
            if (dot_prod > 1.0)
                dot_prod = 1.0;
            shr_pot[i] += tan(abs(M_PI_2 - acos(dot_prod)));
        }

        if (node_info[i+coords_cols*7]==1)
        {
            double dot_prod = b[0]*c[0] + b[1]*c[1] + b[2]*c[2];
            if (dot_prod < -1.0)
                dot_prod = -1.0;
            if (dot_prod > 1.0)
                dot_prod = 1.0;
            shr_pot[i] += tan(abs(M_PI_2 - acos(dot_prod)));
        }
        
        if (node_info[i+coords_cols*8]==1)
        {
            double dot_prod = b[0]*d[0] + b[1]*d[1] + b[2]*d[2];
            if (dot_prod < -1.0)
                dot_prod = -1.0;
            if (dot_prod > 1.0)
                dot_prod = 1.0;
            shr_pot[i] += tan(abs(M_PI_2 - acos(dot_prod)));
        }
        
        if (node_info[i+coords_cols*9]==1)
        {
            double dot_prod = a[0]*d[0] + a[1]*d[1] + a[2]*d[2];
            if (dot_prod < -1.0)
                dot_prod = -1.0;
            if (dot_prod > 1.0)
                dot_prod = 1.0;
            shr_pot[i] += tan(abs(M_PI_2 - acos(dot_prod)));
        }

        // Bending Energy
        if (node_info[i+coords_cols*4]==1)
        {
            double dot_prod = a[0]*b[0] + a[1]*b[1] + a[2]*b[2];
            if (dot_prod < -1.0)
                dot_prod = -1.0;
            if (dot_prod > 1.0)
                dot_prod = 1.0;
//             bndg_pot[i] += tan((M_PI - acos(dot_prod))/2);
            angle = acos(dot_prod);
            bndg_pot[i] += 1.124e10 * exp(-15.34 * angle) + 0.06109 * exp(-1.1386 * angle);
        }
        
        if (node_info[i+coords_cols*5]==1)
        {
            double dot_prod = d[0]*c[0] + d[1]*c[1] + d[2]*c[2];
            if (dot_prod < -1.0)
                dot_prod = -1.0;
            if (dot_prod > 1.0)
                dot_prod = 1.0;
//             bndg_pot[i] += tan((M_PI - acos(dot_prod))/2);
            angle = acos(dot_prod);
            bndg_pot[i] += 1.124e10 * exp(-15.34 * angle) + 0.06109 * exp(-1.1386 * angle);
        }
    }

}


namespace opt_obj
{
    // defualt nlopt fuction
    double customminfunc(const std::vector<double>& x, std::vector<double>& grad, void* data) {
        // Because we wanted a Class
        // without static members, but NLOpt library does not support
        // passing methods of Classes, we use these auxilary functions.
        opt_obj *c = (opt_obj *) data;
        return c->ObjFun(x,grad);
    }

    // class constructor
    opt_obj::opt_obj( Eigen::MatrixXd _coords, Eigen::MatrixXd _prop, int _NoParticles, Eigen::MatrixXd _freeIndices, 
        std::vector<double> _X_init, double OptH, double OptXtolRel, double node_info_arr[], int _node_info_size) 
    {

        //choose optimizer
        // alg_type = nlopt::LN_NEWUOA;
        // optalg = nlopt::LN_NEWUOA_BOUND;
        // alg_type = nlopt::LN_BOBYQA;
        // alg_type = nlopt::LN_COBYLA;
        // alg_type = nlopt::LD_SLSQP;
        alg_type = nlopt::LD_LBFGS;
        // optalg = nlopt::GN_ISRES;
        
        
        coords = _coords;
        properties = _prop;
        node_info_size = _node_info_size;
        no_particles = _NoParticles;
        free_indices = _freeIndices;
        X_init = _X_init;
        optH = OptH;
        optXtolRel = OptXtolRel;
        solminf = 0;
        ptr_node_info_arr = &node_info_arr[0];

        // Optimization Parameters
        OptVarDim = _X_init.size();
        opt = nlopt::opt(alg_type, OptVarDim);
        // OptVarlb.resize(OptVarDim);
        // OptVarub.resize(OptVarDim);
        
        // for (int i=0;i<OptVarlb.size();++i)
        // {
        //     OptVarlb[i] = -100000;
        //     OptVarub[i] = 100000;
        // }

        opt.set_xtol_rel(optXtolRel);
        opt.set_min_objective(customminfunc, this);

        // opt.set_lower_bounds(OptVarlb);
        // opt.set_upper_bounds(OptVarub);
        opt.set_maxeval(50);
        
        // optimization params
        // OptVarDim = x_start.size();

        
        // opt.set_xtol_rel(optXtolRel);
        optx.resize(OptVarDim);
        // opt.set_min_objective(customminfunc, this);
    }

    // class destructor
    opt_obj::~opt_obj()
    {
    }

    
    double opt_obj::ErrFun_cuda(const std::vector<double> &x)
    {
        int counter = -1;
        for ( int i = 0; i < free_indices.rows(); i++)
        {
            coords(free_indices(i,0),0) = x[counter+=1];
            coords(free_indices(i,0),1) = x[counter+=1];
            coords(free_indices(i,0),2) = x[counter+=1];
        }
        
        
        hipMemcpy(ptr_coords, coords.data(), coords.rows()*coords.cols()*sizeof(double),hipMemcpyHostToDevice);
        
        energy_calc <<< block_count+1, no_particles%1000 >>> (ptr_node_info, ptr_coords, ptr_spr_pot, ptr_shr_pot, ptr_bndg_pot, properties(4,0), coords.rows());
        
        
        // auto gpu_start = std::chrono::high_resolution_clock::now();
        hipMemcpy(shr_pot.data(), ptr_shr_pot, no_particles*sizeof(double),hipMemcpyDeviceToHost);
        hipMemcpy(spr_pot.data(), ptr_spr_pot, no_particles*sizeof(double),hipMemcpyDeviceToHost);
        hipMemcpy(bndg_pot.data(), ptr_bndg_pot, no_particles*sizeof(double),hipMemcpyDeviceToHost);
        // auto gpu_end = std::chrono::high_resolution_clock::now();
        // std::cout << " time 2 : " << std::chrono::duration_cast<std::chrono::microseconds>(gpu_end - gpu_start).count() << " microseconds.\n";
        
        std::cout << properties(0,0)*coords.col(2).array().abs().sum() + properties(1,0)*spr_pot.sum() + properties(2,0)*shr_pot.sum() + properties(3,0)*bndg_pot.sum() << std::endl;
        return properties(0,0)*coords.col(2).array().abs().sum() + properties(1,0)*spr_pot.sum() + properties(2,0)*shr_pot.sum() + properties(3,0)*bndg_pot.sum();
        
    }

    bool opt_obj::solveOPT()
    {
        spr_pot = Eigen::MatrixXd::Constant(no_particles,1,0);
        shr_pot = Eigen::MatrixXd::Constant(no_particles,1,0);
        bndg_pot = Eigen::MatrixXd::Constant(no_particles,1,0);
        
        hipMalloc((void**)&ptr_spr_pot, no_particles*1*sizeof(double));
        hipMalloc((void**)&ptr_shr_pot, no_particles*1*sizeof(double));
        hipMalloc((void**)&ptr_bndg_pot, no_particles*1*sizeof(double));
        hipMalloc((void**)&ptr_coords, coords.rows()*coords.cols()*sizeof(double));
        
        hipMalloc((void**)&ptr_node_info, node_info_size*sizeof(double));
        hipMemcpy(ptr_node_info, ptr_node_info_arr, node_info_size*sizeof(double),hipMemcpyHostToDevice);
        
        block_count = no_particles/1000;
        solx = X_init;
        bool successFlag = false;
        try
        {    
            nlopt::result result = opt.optimize(solx, solminf);
            successFlag = true;
        }
        catch(std::exception &e) {
            std::cout << "nlopt failed: " << e.what() << std::endl;
        }

        hipFree(ptr_node_info);
        hipFree(ptr_spr_pot);
        hipFree(ptr_shr_pot);
        hipFree(ptr_bndg_pot);
        hipFree(ptr_coords);
        
        return successFlag;
    };

    std::vector<double> opt_obj::get_solx()
    {
        return solx;
    };

    double opt_obj::get_solminf()
    {
        return solminf;
    };

    // gradient computation:
    // Forward Difference Method
    double opt_obj::ObjFun(const std::vector<double> &x, std::vector<double> &grad)
    {
        double err = ErrFun_cuda(x);
        if (!grad.empty()) {
            std::vector<double> xph = x;
            for (uint i=0; i < x.size(); ++i)
            {
                xph[i] += optH;
                grad[i] = (ErrFun_cuda(xph)-err)/optH;
                xph[i] -= optH;
            }
        }    
        return err;
    };
}

